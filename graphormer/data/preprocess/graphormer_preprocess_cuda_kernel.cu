#include "hip/hip_runtime.h"
#include "graphormer_preprocess.hpp"

#define FLOYD_WARSHALL_BLOCK_DIM_X (16)
#define FLOYD_WARSHALL_BLOCK_DIM_Y (16)

template <typename int_t>
__global__ void floyd_warshall_set_init_kernel(
  int_t num_nodes,
  int_t max_dist,
  const torch::PackedTensorAccessor64<int_t, 2> adj,
  torch::PackedTensorAccessor64<int_t, 2> output_dist) {
  const int_t i = static_cast<int_t>(threadIdx.x + blockIdx.x * blockDim.x);
  const int_t j = static_cast<int_t>(threadIdx.y + blockIdx.y * blockDim.y);
  if (i < num_nodes && j < num_nodes) {
    if (i == j) {
      output_dist[i][j] = 0;
    } else {
      if (adj[i][j]) {
        output_dist[i][j] = 1;
      } else {
        output_dist[i][j] = max_dist;
      }
    }
  }
}

template <typename int_t>
__global__ void floyd_warshall_cuda_one_iter_kernel(
  int_t num_nodes,
  torch::PackedTensorAccessor64<int_t, 2> output_dist,
  torch::PackedTensorAccessor64<int_t, 2> output_pred,
  int_t k) {
  const int_t i = static_cast<int_t>(threadIdx.x + blockIdx.x * blockDim.x);
  const int_t j = static_cast<int_t>(threadIdx.y + blockIdx.y * blockDim.y);
  if (i < num_nodes && j < num_nodes && i != k && j != k && i != j) {
    const int_t new_dist = output_dist[i][k] + output_dist[k][j];
    const int_t old_dist = output_dist[i][j];
    if (new_dist < old_dist) {
      output_dist[i][j] = new_dist;
      output_pred[i][j] = k;
    }
  }
}

std::vector<torch::Tensor> floyd_warshall_cuda(const torch::Tensor adj, const size_t max_dist) {
  const size_t num_nodes = adj.size(0);
  const size_t grid_dim_x = (num_nodes + FLOYD_WARSHALL_BLOCK_DIM_X - 1) / FLOYD_WARSHALL_BLOCK_DIM_X;
  const size_t grid_dim_y = (num_nodes + FLOYD_WARSHALL_BLOCK_DIM_Y - 1) / FLOYD_WARSHALL_BLOCK_DIM_Y;
  dim3 grid_dim(grid_dim_x, grid_dim_y), block_dim(FLOYD_WARSHALL_BLOCK_DIM_X, FLOYD_WARSHALL_BLOCK_DIM_Y);
  auto output_dist = torch::zeros_like(adj);
  auto output_pred = torch::zeros_like(adj);
  AT_DISPATCH_INTEGRAL_TYPES(
    adj.type(), "floyd_warshall_set_init_kernel", ([&] {
      floyd_warshall_set_init_kernel<scalar_t><<<grid_dim, block_dim>>>(
        num_nodes,
        max_dist,
        adj.packed_accessor64<scalar_t, 2>(),
        output_dist.packed_accessor64<scalar_t, 2>());
  }));
  for (size_t k = 0; k < num_nodes; ++k) {
    AT_DISPATCH_INTEGRAL_TYPES(
      adj.type(), "floyd_warshall_cuda_one_iter_kernel", ([&] {
        floyd_warshall_cuda_one_iter_kernel<scalar_t><<<grid_dim, block_dim>>>(
          num_nodes,
          output_dist.packed_accessor64<scalar_t, 2>(),
          output_pred.packed_accessor64<scalar_t, 2>(),
          k);
    }));
  }
  return {output_dist, output_pred};
}

template <typename int_t, size_t MAX_DIST>
__device__ void get_path(
  const int_t num_nodes,
  const int_t max_dist,
  const int_t i,
  const int_t j,
  const torch::PackedTensorAccessor64<int_t, 2> pred,
  const torch::PackedTensorAccessor64<int_t, 2> dist,
  int_t* out_path) {
  int_t stk[MAX_DIST * 2 + 10];
  int64_t stk_ptr = 0;
  stk[0] = i;
  stk[1] = j;
  stk[2] = max_dist;
  stk[3] = max_dist;
  while (stk_ptr >= 0) {
    const int_t stk_i = stk[2 * stk_ptr];
    const int_t stk_j = stk[2 * stk_ptr + 1];
    const int_t next_flag = stk[2 * stk_ptr + 2];

    const int_t stk_k = pred[stk_i][stk_j];
    const int_t stk_i_k_dist = dist[stk_i][stk_k];

    if (next_flag == max_dist) {
      // push first segment
      out_path[stk_i_k_dist - 1] = stk_k;

      ++stk_ptr;
      stk[2 * stk_ptr] = stk_i;
      stk[2 * stk_ptr + 1] = stk_k;
      if (stk_i_k_dist > 1) {
        stk[2 * (stk_ptr + 1)] = max_dist;
        stk[2 * (stk_ptr + 1) + 1] = max_dist;
      } else {
        --stk_ptr;
      }
    } else if (next_flag == stk_i) {
      // push second segment
      const int_t stk_k_j_dist = dist[stk_k][stk_j];
      ++stk_ptr;
      stk[2 * stk_ptr] = stk_k;
      stk[2 * stk_ptr + 1] = stk_j;
      if (stk_k_j_dist > 1) {
        stk[2 * (stk_ptr + 1)] = max_dist;
        stk[2 * (stk_ptr + 1) + 1] = max_dist;
      } else {
        --stk_ptr;
      }
      out_path += stk_i_k_dist;
    } else if (next_flag == stk_k) {
      // callback
      out_path -= stk_i_k_dist;
      --stk_ptr;
    }
  }
}

template <typename int_t, size_t MAX_DIST>
__global__ void gen_edge_input_kernel(
  const int_t num_nodes,
  const int_t max_dist,
  const torch::PackedTensorAccessor64<int_t, 2> pred,
  const torch::PackedTensorAccessor64<int_t, 2> dist,
  const int_t num_edge_features,
  const torch::PackedTensorAccessor64<int_t, 3> edge_features,
  torch::PackedTensorAccessor64<int_t, 4> output_edge_features) {
  const int i = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
  const int j = static_cast<int>(threadIdx.y + blockIdx.y * blockDim.y);
  if (i < num_nodes && j < num_nodes) {
    int_t path[MAX_DIST];
    const int_t i_j_dist = dist[i][j];
    if (i_j_dist > 1 && i_j_dist != max_dist) {
      get_path<int_t, MAX_DIST>(num_nodes, max_dist, i, j, pred, dist, path);
    }
    if (i_j_dist != max_dist) {
      int start = i;
      for (int e = 0; e < i_j_dist; ++e) {
        const int end = (e == i_j_dist - 1 ? j : path[e]);
        for (int feature_index = 0; feature_index < num_edge_features; ++feature_index) {
          output_edge_features[i][j][e][feature_index] = edge_features[start][end][feature_index];
        }
        start = end;
      }
    }
  }
}

void gen_edge_input_cuda(
  const size_t max_dist,
  const torch::Tensor pred,
  const torch::Tensor dist,
  const size_t num_edge_features,
  const torch::Tensor edge_features,
  torch::Tensor output_edge_features) {
  const size_t num_nodes = pred.size(0);
  const size_t grid_dim_x = (num_nodes + FLOYD_WARSHALL_BLOCK_DIM_X - 1) / FLOYD_WARSHALL_BLOCK_DIM_X;
  const size_t grid_dim_y = (num_nodes + FLOYD_WARSHALL_BLOCK_DIM_Y - 1) / FLOYD_WARSHALL_BLOCK_DIM_Y;
  dim3 grid_dim(grid_dim_x, grid_dim_y), block_dim(FLOYD_WARSHALL_BLOCK_DIM_X, FLOYD_WARSHALL_BLOCK_DIM_Y);
  AT_DISPATCH_INTEGRAL_TYPES(pred.type(), "gen_edge_input_kernel", ([&] {
    auto dist_accessor = dist.packed_accessor64<scalar_t, 2>();
    auto pred_accessor = pred.packed_accessor64<scalar_t, 2>();
    auto edge_features_accessor = edge_features.packed_accessor64<scalar_t, 3>();
    auto output_edge_features_accessor = output_edge_features.packed_accessor64<scalar_t, 4>();
    if (max_dist <= 16) {
      gen_edge_input_kernel<scalar_t, 16><<<grid_dim, block_dim>>>(
        num_nodes, max_dist, pred_accessor, dist_accessor, num_edge_features, edge_features_accessor, output_edge_features_accessor);
    } else if (max_dist <= 32) {
      gen_edge_input_kernel<scalar_t, 32><<<grid_dim, block_dim>>>(
        num_nodes, max_dist, pred_accessor, dist_accessor, num_edge_features, edge_features_accessor, output_edge_features_accessor);
    } else if (max_dist <= 64) {
      gen_edge_input_kernel<scalar_t, 64><<<grid_dim, block_dim>>>(
        num_nodes, max_dist, pred_accessor, dist_accessor, num_edge_features, edge_features_accessor, output_edge_features_accessor);
    } else if (max_dist <= 128) {
      gen_edge_input_kernel<scalar_t, 128><<<grid_dim, block_dim>>>(
        num_nodes, max_dist, pred_accessor, dist_accessor, num_edge_features, edge_features_accessor, output_edge_features_accessor);
    } else if (max_dist <= 256) {
      gen_edge_input_kernel<scalar_t, 256><<<grid_dim, block_dim>>>(
        num_nodes, max_dist, pred_accessor, dist_accessor, num_edge_features, edge_features_accessor, output_edge_features_accessor);
    } else if (max_dist <= 512) {
      gen_edge_input_kernel<scalar_t, 512><<<grid_dim, block_dim>>>(
        num_nodes, max_dist, pred_accessor, dist_accessor, num_edge_features, edge_features_accessor, output_edge_features_accessor);
    } else {
      std::cout << "error, max_dist = " << max_dist << " is not supported by gen_edge_input." << std::endl;
      exit(-1);
    }
  }));
}
